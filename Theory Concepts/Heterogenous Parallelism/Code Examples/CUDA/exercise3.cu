#include "hip/hip_runtime.h"
/* Udacity Homework 3
HDR Tone-mapping
Background HDR
==============
A High Dynamic Range (HDR) image contains a wider variation of intensity
and color than is allowed by the RGB format with 1 byte per channel that we
have used in the previous assignment.
To store this extra information we use single precision floating point for
each channel.  This allows for an extremely wide range of intensity values.
In the image for this assignment, the inside of church with light coming in
through stained glass windows, the raw input floating point values for the
channels range from 0 to 275.  But the mean is .41 and 98% of the values are
less than 3!  This means that certain areas (the windows) are extremely bright
compared to everywhere else.  If we linearly map this [0-275] range into the
[0-255] range that we have been using then most values will be mapped to zero!
The only thing we will be able to see are the very brightest areas - the
windows - everything else will appear pitch black.
The problem is that although we have cameras capable of recording the wide
range of intensity that exists in the real world our monitors are not capable
of displaying them.  Our eyes are also quite capable of observing a much wider
range of intensities than our image formats / monitors are capable of
displaying.
Tone-mapping is a process that transforms the intensities in the image so that
the brightest values aren't nearly so far away from the mean.  That way when
we transform the values into [0-255] we can actually see the entire image.
There are many ways to perform this process and it is as much an art as a
science - there is no single "right" answer.  In this homework we will
implement one possible technique.
Background Chrominance-Luminance
================================
The RGB space that we have been using to represent images can be thought of as
one possible set of axes spanning a three dimensional space of color.  We
sometimes choose other axes to represent this space because they make certain
operations more convenient.
Another possible way of representing a color image is to separate the color
information (chromaticity) from the brightness information.  There are
multiple different methods for doing this - a common one during the analog
television days was known as Chrominance-Luminance or YUV.
We choose to represent the image in this way so that we can remap only the
intensity channel and then recombine the new intensity values with the color
information to form the final image.
Old TV signals used to be transmitted in this way so that black & white
televisions could display the luminance channel while color televisions would
display all three of the channels.
Tone-mapping
============
In this assignment we are going to transform the luminance channel (actually
the log of the luminance, but this is unimportant for the parts of the
algorithm that you will be implementing) by compressing its range to [0, 1].
To do this we need the cumulative distribution of the luminance values.
Example
-------
input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
min / max / range: 0 / 9 / 9
histo with 3 bins: [4 7 3]
// exclusive
cdf : [0 4 11]
Your task is to calculate this cumulative distribution by following these
steps.
*/

#include "utils.h"
#include<>
#include<hip/device_functions.h>

__global__ void shmem_reduce_kernel(float * d_out, const float * const d_in, bool is_max)
{
  // sdata is allocated in the kernel call: 3rd arg to <<<b, t, shmem>>>
  extern __shared__ float sdata[];

  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int tid = threadIdx.x;

  // load shared mem from global mem
  sdata[tid] = d_in[myId];
  __syncthreads();            // make sure entire block is loaded!

  // do reduction in shared mem
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
  {
    if (tid < s)
    {
      if (is_max)
        sdata[tid] = max(sdata[tid], sdata[tid + s]);
      else
        sdata[tid] = min(sdata[tid], sdata[tid + s]);
    }
    __syncthreads();        // make sure all adds at one stage are done!
  }

  // only thread 0 writes result for this block back to global mem
  if (tid == 0)
  {
    d_out[blockIdx.x] = sdata[0];
  }
}

__global__ void histo_kernel(unsigned int * d_out, const float * const d_in,
  const size_t numBins, float logLumRange, float min_logLum)
{
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int bin = (d_in[myId] - min_logLum) / logLumRange * numBins;
  if (bin == numBins)  bin--;
  atomicAdd(&d_out[bin], 1);
}

__global__ void scan_kernel(unsigned int * d_out, const float * const d_in,
  const size_t numBins, float logLumRange, float min_logLum)
{
  int myId = threadIdx.x + blockDim.x * blockIdx.x;
  int bin = (d_in[myId] - min_logLum) / logLumRange * numBins;
  if (bin == numBins)  bin--;
  atomicAdd(&d_out[bin], 1);
}

// Hillis Steele Scan - described in lecture
__global__ void cdf_kernel(unsigned int * d_in, const size_t numBins)
{
  int myId = threadIdx.x;
  for (int d = 1; d < numBins; d *= 2) {
    if ((myId + 1) % (d * 2) == 0) {
      d_in[myId] += d_in[myId - d];
    }
    __syncthreads();
  }
  if (myId == numBins - 1) d_in[myId] = 0;
  for (int d = numBins / 2; d >= 1; d /= 2) {
    if ((myId + 1) % (d * 2) == 0) {
      unsigned int tmp = d_in[myId - d];
      d_in[myId - d] = d_in[myId];
      d_in[myId] += tmp;
    }
    __syncthreads();
  }
}

// Blelloch Scan - described in lecture
__global__ void cdf_kernel_2(unsigned int * d_in, const size_t numBins)
{ 
  int idx = threadIdx.x;
  extern __shared__ int temp[];
  int pout = 0, pin = 1;

  temp[idx] = (idx > 0) ? d_in[idx - 1] : 0;
  __syncthreads();

  for (int offset = 1; offset < n; offset *= 2) {
    // swap double buffer indices
    pout = 1 - pout;
    pin = 1 - pout;
    if (idx >= offset) {
      temp[pout*n+idx] = temp[pin*n+idx - offset] + temp[pin*n+idx];  // changed line
    } else {
      temp[pout*n+idx] = temp[pin*n+idx];
    }
    __syncthreads();
  }
  d_in[idx] = temp[pout*n+idx];
}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
  unsigned int* const d_cdf,
  float &min_logLum,
  float &max_logLum,
  const size_t numRows,
  const size_t numCols,
  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
  1) find the minimum and maximum value in the input logLuminance channel
  store in min_logLum and max_logLum */
  // use reduce
  const int m = 1 << 10;
  int blocks = ceil((float)numCols * numRows / m);

  float *d_intermediate; // should not modify d_in
  checkCudaErrors(hipMalloc(&d_intermediate, sizeof(float)* blocks)); // store max and min
  float *d_min, *d_max;
  checkCudaErrors(hipMalloc((void **)&d_min, sizeof(float)));
  checkCudaErrors(hipMalloc((void **)&d_max, sizeof(float)));

  shmem_reduce_kernel << <blocks, m, m * sizeof(float) >> >(d_intermediate, d_logLuminance, true);
  shmem_reduce_kernel << <1, blocks, blocks * sizeof(float) >> >(d_max, d_intermediate, true);
  shmem_reduce_kernel << <blocks, m, m * sizeof(float) >> >(d_intermediate, d_logLuminance, false);
  shmem_reduce_kernel << <1, blocks, blocks * sizeof(float) >> >(d_min, d_intermediate, false);
  checkCudaErrors(hipMemcpy(&min_logLum, d_min, sizeof(float), hipMemcpyDeviceToHost));
  checkCudaErrors(hipMemcpy(&max_logLum, d_max, sizeof(float), hipMemcpyDeviceToHost));

  checkCudaErrors(hipFree(d_intermediate));
  checkCudaErrors(hipFree(d_min));
  checkCudaErrors(hipFree(d_max));

  //2) subtract them to find the range
  float logLumRange = max_logLum - min_logLum;
  printf("max_logLum: %f  min_logLum: %f  logLumRange: %f\n", max_logLum, min_logLum, logLumRange);

  //3) generate a histogram of all the values in the logLuminance channel using
  //   the formula: bin = (lum[i] - lumMin) / lumRange * numBins
  checkCudaErrors(hipMemset(d_cdf, 0, sizeof(unsigned int)* numBins));
  histo_kernel << <blocks, m >> >(d_cdf, d_logLuminance, numBins, logLumRange, min_logLum);

  //4) Perform an exclusive scan (prefix sum) on the histogram to get
  //   the cumulative distribution of luminance values (this should go in the
  //   incoming d_cdf pointer which already has been allocated for you)      
  //cdf_kernel << <1, numBins>> >(d_cdf, numBins);
  cdf_kernel_2 << <1, numBins, sizeof(unsigned int) * numBins * 2 >> >(d_cdf, numBins);
}